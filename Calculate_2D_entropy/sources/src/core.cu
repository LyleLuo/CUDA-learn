#include "hip/hip_runtime.h"
#include "core.h"

__global__ void kernel(int width, int height, float *input, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height) {
        int cnt[16] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
        for (int pos_y = idy - 2; pos_y <= idy + 2; ++pos_y) {
            if (pos_y >= 0 && pos_y < height) {
                for (int pos_x = idx - 2; pos_x <= idx + 2; ++pos_x) {
                    if (pos_x >= 0 && pos_x < width) {
                        cnt[(int)input[pos_x + pos_y*width]]++;
                    }
                }
            }
        }
        
        double n = (min(idx, 2) + 1 + min(width - idx, 2)) * (min(idy, 2) + 1 + min(height - idy, 2));
        double n_inv = 1.0 / n;
        double ans = log(n);

        for (int i = 0; i < 16; ++i) {
            if (cnt[i]) {
                ans -= log((double)cnt[i]) * cnt[i] * n_inv;
            }
        }

        output[idy * width + idx] = ans;
    }
}

void cudaCallback(int width, int height, float *sample, float **result) {
    int size = width * height;
    float *input_d, *output_d;

    // Allocate device memory and copy data from host to device
    CHECK(hipMalloc((void **)&input_d, sizeof(float)*size));
    CHECK(hipMalloc((void **)&output_d, sizeof(float)*size));
    CHECK(hipMemcpy(input_d, sample, sizeof(float)*size, hipMemcpyHostToDevice));

    // Invoke the device function

    const dim3 blockDim(32, 32), gridDim(divup(width, 32), divup(height, 32));

    kernel<<< gridDim, blockDim >>>(width, height, input_d, output_d);
    hipDeviceSynchronize();

    // Copy back the results and de-allocate the device memory
    *result = (float *)malloc(sizeof(float)*size);
    CHECK(hipMemcpy(*result, output_d, sizeof(float)*size, hipMemcpyDeviceToHost));
    CHECK(hipFree(input_d));
    CHECK(hipFree(output_d))
    
    // Note that you don't have to free sample and *result by yourself
}
